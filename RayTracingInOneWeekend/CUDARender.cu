#include "hip/hip_runtime.h"
#include "pch.cuh"

#include <hip/hip_runtime.h>
#include <future>
#include <hip/hip_runtime.h>
#include "CudaRenderer.cuh"

#include <hiprand/hiprand_kernel.h>
#include <mutex>

#include "BVH.h"
#include "CudaCamera.cuh"
#include "HittableList.h"
#include "Random.h"
#include "Material.h"
#include "Sphere.h"

#define RND (hiprand_uniform(&local_rand_state))

__host__ void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		// exit(99);
	}
}

__global__ void rand_init(hiprandState* rand_state)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		hiprand_init(1984, 0, 0, rand_state);
	}
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y))
		return;
	int pixel_index = j * max_x + i;
	// Original: Each thread gets same seed, a different sequence number, no offset
	// hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
	// BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
	// performance improvement of about 2x!
	hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void create_world(Sphere* d_spheres, Hittable** d_world, int nx, int ny, hiprandState* rand_state)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		hiprandState local_rand_state = *rand_state;
		int			i				 = 0;

		// Ground sphere:
		new (&d_spheres[i++]) Sphere(vec3(0, -1000.0f, -1),
									 1000.0f,
									 Material(MaterialType::Lambert, vec3(0.5f, 0.5f, 0.5f)));

		// For each grid position:
		for (int a = -11; a < 11; a++)
		{
			for (int b = -11; b < 11; b++)
			{
				float choose_mat = RND;
				vec3  center(a + RND, 0.2f, b + RND);
				if (choose_mat < 0.8f)
				{
					// Create a Lambertian sphere
					new (&d_spheres[i++]) Sphere(center,
												 0.2f,
												 Material(MaterialType::Lambert, vec3(RND * RND, RND * RND, RND * RND)));
				}
				else if (choose_mat < 0.95f)
				{
					// Create a u_Metal sphere
					new (&d_spheres[i++]) Sphere(center,
												 0.2f,
												 Material(MaterialType::Metal, vec3(0.5f * (1 + RND), 0.5f * (1 + RND), 0.5f * (1 + RND)), 0.5f * RND));
				}
				else
				{
					// Create a u_Dielectric sphere
					new (&d_spheres[i++]) Sphere(center,
												 0.2f,
												 Material(MaterialType::Dielectric, 1.0, 0.0f, 1.5f));
				}
			}
		}

		// Add the three big spheres:
		new (&d_spheres[i++]) Sphere(vec3(0, 1, 0),
									 1.0f,
									 Material(MaterialType::Dielectric, 1.0, 0.0f, 1.5f));

		new (&d_spheres[i++]) Sphere(vec3(-4, 1, 0),
									 1.0f,
									 Material(MaterialType::Lambert, vec3(0.4f, 0.2f, 0.1f)));

		new (&d_spheres[i++]) Sphere(vec3(4, 1, 0),
									 1.0f,
									 Material(MaterialType::Metal, vec3(0.7f, 0.6f, 0.5f), 0.0f));

		*rand_state = local_rand_state;

		Hittable** spherePtrs = new Hittable*[i];
		for (int j = 0; j < i; j++)
		{
			spherePtrs[j] = reinterpret_cast<Hittable*>(&d_spheres[j]);
		}
		*d_world = new BVHNode(HittableList(spherePtrs, i), 0.0, 1.0, &local_rand_state);
		delete[] spherePtrs;
	}
}

//__device__ vec3 unit_vector(const vec3& v)
//{
//	float length = sqrtf(v.x() * v.x() + v.y() * v.y() + v.z() * v.z());
//	return vec3(v.x() / length, v.y() / length, v.z() / length);
//}

[[nodiscard]] __device__ vec3 RayColor(Ray ray, Hittable** world, const uint32_t depth, hiprandState* local_rand_state)
{
	Ray	 cur_ray		 = ray;
	vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
	for (uint32_t i = 0; i < depth; i++)
	{
		HitRecord rec;
		if ((*world)->Hit(cur_ray, 0.001f, FLT_MAX, rec))
		{
			Ray	 scattered;
			vec3 attenuation;
			if (rec.MaterialPtr->Scatter(cur_ray, rec, attenuation, scattered, local_rand_state))
			{
				cur_attenuation *= attenuation;
				cur_ray = scattered;

				//// Russian Roulette only in shadows
				//if (cur_attenuation.x() < 0.001f || cur_attenuation.y() < 0.001f || cur_attenuation.z() < 0.001f)
				//{
				//	float rrPcont = (std::max(cur_attenuation.x(), std::max(cur_attenuation.y(), cur_attenuation.z())) + 0.001f);

				//	if (hiprand_uniform(local_rand_state) > rrPcont)
				//		break; // Terminate the path

				//	cur_attenuation /= rrPcont; // Adjust throughput for Russian Roulette
				//}
			}
			else
			{
				return vec3(0.0, 0.0, 0.0);
			}
		}
		else
		{
			vec3  unit_direction = unit_vector(cur_ray.Direction());
			float t				 = 0.5f * (unit_direction.y() + 1.0f);
			vec3  c				 = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	return {0.0, 0.0, 0.0}; // exceeded recursion
}

__global__ void InternalRender(vec3* fb, Hittable** world, uint32_t max_x, uint32_t max_y, Camera* camera, uint32_t samplersPerPixel, float colorMul, uint32_t maxDepth, hiprandState* rand_state)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y))
		return;

	int pixel_index = j * max_x + i;

	hiprandState& local_rand_state = rand_state[pixel_index];
	vec3		 pixel_color(0.0f);
	for (uint32_t s = 0; s < samplersPerPixel; s++)
	{
		float u	 = float(float(i) + hiprand_uniform(&local_rand_state)) / float(max_x);
		float v	 = float(float(j) + hiprand_uniform(&local_rand_state)) / float(max_y);
		u		 = 1.0f - u;
		v		 = 1.0f - v;
		auto ray = camera->GetRay(u, v);

		pixel_color += RayColor(ray, world, maxDepth, &local_rand_state);
	}
	rand_state[pixel_index] = local_rand_state;
	pixel_color *= colorMul;
	pixel_color		= vec3(sqrt(pixel_color.x()), sqrt(pixel_color.y()), sqrt(pixel_color.z()));
	fb[pixel_index] = pixel_color;
}

__host__ void CudaRenderer::Init()
{
	hipDeviceSetLimit(hipLimitStackSize, 4096);

	// allocate random state
	checkCudaErrors(hipMalloc((void**)&d_rand_state, m_Width * m_Height * sizeof(hiprandState)));
	checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

	int num_hitables = 22 * 22 + 1 + 3;
	checkCudaErrors(hipMallocManaged((void**)&d_list, num_hitables * sizeof(Sphere)));
	checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Hittable*)));
	checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(Camera)));

	rand_init<<<1, 1>>>(d_rand_state2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	uint32_t threads = 8;
	dim3	 blocks(m_Width / threads + 1, m_Height / threads + 1);
	dim3	 workGroup(threads, threads, 1);

	render_init<<<blocks, workGroup>>>(m_Width, m_Height, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	create_world<<<1, 1>>>((Sphere*)d_list, d_world, m_Width, m_Height, d_rand_state2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

__host__ void CudaRenderer::Render() const
{
	uint32_t threads = 16;
	dim3	 blocks(m_Width / threads + 1, m_Height / threads + 1);
	dim3	 workGroup(threads, threads, 1);
	float	 aspectRatio = float(m_Width) / float(m_Height);

	static float distance = 0.0f;
	Camera		 camera(vec3(13.0f + distance, 2.0f, 3.0f), vec3(0.0f, 0.0f, 0.0f), vec3(0.0f, 1.0f, 0.0f), 20.0f, aspectRatio);

	checkCudaErrors(hipMemcpy(d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice));
	checkCudaErrors(hipDeviceSynchronize());
	distance += 0.1f;

	if (distance > 10.0f)
		distance = 0.0f;

	clock_t start, stop;
	start = clock();
	// Render our buffer
	InternalRender<<<blocks, workGroup>>>(d_Image, d_world, m_Width, m_Height, d_camera, m_SamplesPerPixel, m_ColorMul, m_MaxDepth, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	stop				 = clock();
	double timer_seconds = ((double)(stop - start));
	std::cerr << "took " << timer_seconds << "ms.\n";
}

__host__ std::vector<float> CudaRenderer::CopyImage()
{
	std::vector<float> h_Pixels(m_Width * m_Height * 3 * sizeof(float));
	hipMemcpy(h_Pixels.data(), d_Image, h_Pixels.size(), hipMemcpyDeviceToHost);

	return h_Pixels;
}
