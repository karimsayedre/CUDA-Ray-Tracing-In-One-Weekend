#include "hip/hip_runtime.h"
#include "pch.cuh"
#include "BVH.h"
#include "HittableList.h"
#include "Material.h"
#include "Sphere.h"
#include "Random.h"

#define RND (hiprand_uniform(&local_rand_state))

__host__ void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
	if (result)
	{
		printf("CUDA error = %s at %s:%d '%s' \n", hipGetErrorString(result), file, line, func);
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__global__ void rand_init(hiprandState* rand_state)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		hiprand_init(1984, 0, 0, rand_state);
	}
}

__global__ void render_init(uint32_t max_x, uint32_t max_y, uint32_t* rand_state)
{
	uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y))
		return;
	uint32_t pixel_index = j * max_x + i;
	// Original: Each thread gets same seed, a different sequence number, no offset
	// hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
	// BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
	// performance improvement of about 2x!
	// hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);

	rand_state[pixel_index] = pcg_hash(1984 + pixel_index);
}

__global__ void create_world(HittableList* d_list, Materials* d_materials, BVHSoA* d_world, int nx, int ny, hiprandState* rand_state)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		hiprandState local_rand_state = *rand_state;
		uint16_t	i				 = 0;

		d_materials->Add(MaterialType::Lambert, Vec3(0.5f, 0.5f, 0.5f), 0.0f, 1.0f);
		// Ground sphere:
		new (&d_list->m_Objects[i]) Sphere(Vec3(0, -1000.0f, -1), 1000.0f, i);
		i++;

		// For each grid position:
		for (int a = -11; a < 11; a++)
		{
			for (int b = -11; b < 11; b++)
			{
				Float choose_mat = RND;
				Vec3  center(a + RND, 0.2f, b + RND);
				if (choose_mat < __float2half(0.8f))
				{
					d_materials->Add(MaterialType::Lambert, Vec3(RND * RND, RND * RND, RND * RND), 0.0f, 1.0f);
					new (&d_list->m_Objects[i]) Sphere(center, 0.2f, i);
					i++;
				}
				else if (choose_mat < __float2half(0.95f))
				{
					d_materials->Add(MaterialType::Metal, Vec3(0.5f * (1 + RND), 0.5f * (1 + RND), 0.5f * (1 + RND)), 0.5f * RND, 1.0f);
					new (&d_list->m_Objects[i]) Sphere(center, 0.2f, i);
					i++;
				}
				else
				{
					d_materials->Add(MaterialType::Dielectric, Vec3(1.0), 0.0f, 1.5f);
					new (&d_list->m_Objects[i]) Sphere(center, 0.2f, i);
					i++;
				}
			}
		}

		// Add the three big spheres:
		d_materials->Add(MaterialType::Dielectric, Vec3(1.0), 0.0f, 1.5f);
		new (&d_list->m_Objects[i]) Sphere(Vec3(0, 1, 0), 1.0f, i);
		i++;
		d_materials->Add(MaterialType::Lambert, Vec3(0.4f, 0.2f, 0.1f), 0.0f, 1.0f);
		new (&d_list->m_Objects[i]) Sphere(Vec3(-4, 1, 0), 1.0f, i);
		i++;

		d_materials->Add(MaterialType::Metal, Vec3(0.7f, 0.6f, 0.5f), 0.0f, 1.0f);
		new (&d_list->m_Objects[i]) Sphere(Vec3(4, 1, 0), 1.0f, i);
		i++;

		d_list->SetAABBs();

		*rand_state = local_rand_state;

		uint32_t* indices = (uint32_t*)malloc(d_list->m_Count * sizeof(uint32_t));
		for (uint32_t index = 0; index < d_list->m_Count; ++index)
			indices[index] = index;

		d_world->root = d_world->BuildBVH_SoA(d_list, indices, 0, d_list->m_Count);
		printf("BVH created with %u nodes out of %u allocated\n", d_world->m_count, d_world->m_capacity);

		printf("BVH Root: %u\n", d_world->root);
		// DebugBVHNode(d_world, d_world->root);

		free(indices);
	}
}

__device__ Vec3 RayColor(Ray& ray, BVHSoA* __restrict__ world, HittableList* __restrict__ list, Materials* __restrict__ materials, const uint32_t depth, uint32_t& randSeed)
{
	Vec3 cur_attenuation(1.0f);
	Ray	 current_ray = ray;

	for (uint32_t i = 0; i < depth; i++)
	{
		HitRecord rec;
		// Use current_ray instead of ray
		if (!world->TraverseBVH_SoA(current_ray, 0.001f, FLT_MAX, list, world->root, rec))
		{
			// Sky color calculation
			Vec3 unit_direction = current_ray.Direction();
			// Float	  inv_length	 = rsqrtf(unit_direction.x * unit_direction.x + unit_direction.y * unit_direction.y + unit_direction.z * unit_direction.z);
			// Vec3 unit_direction = unit_direction * inv_length;

			Float t			= (__float2half(0.5f)) * (unit_direction.y + __float2half(1.0f));
			Vec3  sky_color = (__float2half(1.0f) - t) * Vec3(1.0) + t * Vec3(0.5f, 0.7f, 1.0f);
			return cur_attenuation * sky_color;
		}

		// Russian Roulette for path termination
		// if (i > 3)
		{
			Float rrProb = glm::hmax(cur_attenuation.x, glm::hmax(cur_attenuation.y, cur_attenuation.z));
			if (RandomFloat(randSeed) > rrProb)
				break;
			cur_attenuation /= rrProb;
		}

		// Scatter ray with optimized material interaction
		Ray	 scattered_ray;
		Vec3 attenuation {1.0f};
		if (!materials->Scatter(current_ray, scattered_ray, rec, attenuation, randSeed))
			break;

		// Update attenuation and current ray
		cur_attenuation *= attenuation;
		current_ray = scattered_ray;

		// Early termination for very low contribution
		if (fmaxf(cur_attenuation.x, fmaxf(cur_attenuation.y, cur_attenuation.z)) < 0.001f)
			break;
	}

	return Vec3(0.0f); // Exceeded Max depth
}

__global__ void InternalRender(glm::vec<4, sf::Uint8, glm::packed_lowp>* __restrict__ fb, BVHSoA* __restrict__ world, HittableList* __restrict__ list, Materials* __restrict__ materials, uint32_t max_x, uint32_t max_y, Camera* camera, uint32_t samplersPerPixel, Float colorMul, uint32_t maxDepth, uint32_t* randSeeds)
{
	uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y))
		return;

	uint32_t pixel_index = j * max_x + i;

	uint32_t seed = randSeeds[pixel_index];
	Vec3	 pixel_color(0.0f);
	for (uint32_t s = 0; s < samplersPerPixel; s++)
	{
		Float u = Float(Float(i) + RandomFloat(seed)) / Float(max_x);
		Float v = Float(Float(j) + RandomFloat(seed)) / Float(max_y);
		// u		 = 1.0f - u;
		v		 = __float2half(1.0f) - v;
		auto ray = camera->GetRay(u, v);

		pixel_color += RayColor(ray, world, list, materials, maxDepth, seed);
	}
	randSeeds[pixel_index] = seed;
	pixel_color *= colorMul;
	pixel_color		= glm::sqrt(pixel_color);
	fb[pixel_index] = glm::vec<4, sf::Uint8, glm::packed_lowp>(pixel_color.x * __float2half(255.f), pixel_color.y * __float2half(255.f), pixel_color.z * __float2half(255.f), 255);
}

__host__ void CudaRenderer::Init()
{
	hipDeviceSetLimit(hipLimitStackSize, 20000);
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	// allocate random state
	CHECK_CUDA_ERRORS(hipMalloc((void**)&d_rand_seeds, m_Width * m_Height * sizeof(uint32_t)));
	CHECK_CUDA_ERRORS(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

	constexpr int numHitables = 22 * 22 + 1 + 3;

	// Allocate memory for the HittableList struct in device memory
	CHECK_CUDA_ERRORS(hipMalloc((void**)&d_list, sizeof(HittableList)));

	// Allocate memory for the objects array inside HittableList
	void* tempObjects;
	void* tempAABBs;
	CHECK_CUDA_ERRORS(hipMalloc(&tempObjects, numHitables * sizeof(Sphere)));
	CHECK_CUDA_ERRORS(hipMalloc(&tempAABBs, numHitables * sizeof(AABB)));

	// Copy the device pointer for objects into d_list->m_Objects
	CHECK_CUDA_ERRORS(hipMemcpy(&(d_list->m_Objects), &tempObjects, sizeof(Sphere*), hipMemcpyHostToDevice));
	CHECK_CUDA_ERRORS(hipMemcpy(&(d_list->m_AABB), &tempAABBs, sizeof(AABB*), hipMemcpyHostToDevice));

	// Copy the hitable count to the device
	CHECK_CUDA_ERRORS(hipMemcpy(&(d_list->m_Count), &numHitables, sizeof(int), hipMemcpyHostToDevice));

	BVHSoA::Init(d_world, numHitables * 2 - 1);
	Materials::Init(d_materials, numHitables);

	CHECK_CUDA_ERRORS(hipMalloc((void**)&d_camera, sizeof(Camera)));
	// MaterialSoA::Init(numHitables);

	rand_init<<<1, 1>>>(d_rand_state2);
	CHECK_CUDA_ERRORS(hipGetLastError());
	CHECK_CUDA_ERRORS(hipDeviceSynchronize());

	dim3 block(8, 8);
	dim3 grid((m_Width + block.x - 1) / block.x,
			  (m_Height + block.y - 1) / block.y);

	render_init<<<grid, block>>>(m_Width, m_Height, d_rand_seeds);
	CHECK_CUDA_ERRORS(hipGetLastError());
	CHECK_CUDA_ERRORS(hipDeviceSynchronize());

	create_world<<<1, 1>>>(d_list, d_materials, d_world, m_Width, m_Height, d_rand_state2);
	CHECK_CUDA_ERRORS(hipGetLastError());
	CHECK_CUDA_ERRORS(hipDeviceSynchronize());

	hipDeviceSetLimit(hipLimitStackSize, 200);
}

__host__ void CudaRenderer::Render() const
{
	dim3 block(8, 8);
	dim3 grid((m_Width + block.x - 1) / block.x,
			  (m_Height + block.y - 1) / block.y);

	Float aspectRatio = Float(m_Width) / Float(m_Height);

	static Float distance = 0.0f;
	Camera		 camera(Vec3(__float2half(13.0f) + distance, 2.0f, 3.0f), Vec3(0.0f, 0.0f, 0.0f), Vec3(0.0f, 1.0f, 0.0f), 20.0f, aspectRatio);

	CHECK_CUDA_ERRORS(hipMemcpy(d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice));
	CHECK_CUDA_ERRORS(hipDeviceSynchronize());
	// distance += 0.1f;

	if (distance > __float2half(10.0f))
		distance = 0.0f;

	const clock_t start = clock();
	// Render our buffer
	InternalRender<<<grid, block>>>(d_Image, d_world, d_list, d_materials, m_Width, m_Height, d_camera, m_SamplesPerPixel, m_ColorMul, m_MaxDepth, d_rand_seeds);
	CHECK_CUDA_ERRORS(hipGetLastError());
	CHECK_CUDA_ERRORS(hipDeviceSynchronize());

	const clock_t stop		   = clock();
	const double  timerSeconds = stop - start;
	std::cerr << "took " << timerSeconds << "ms.\n";
}

CudaRenderer::~CudaRenderer()
{
	hipFree(d_rand_seeds);
	hipFree(d_rand_state2);
	hipFree(d_list);
	// hipFree(d_materials);
	hipFree(d_world);
	hipFree(d_camera);
	hipFree(d_Image);
}
